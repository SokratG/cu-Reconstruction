#include "hip/hip_runtime.h"
#include "util.cuh"
#include "CudaUtils/cudaUtility.cuh"




namespace cuphoto {

__host__ inline int divUp(int a, int b){
	return ((a % b) != 0) ? (a / b + 1) : (a / b);
}


__global__ void random_float3_kernel(hiprandState* curandstate, const r32 scale, float3& value) {
    const i32 idx = blockIdx.x * blockDim.x + threadIdx.x;
    const r32 randf_x = hiprand_uniform(curandstate + idx) * scale;
    const r32 randf_y = hiprand_uniform(curandstate + idx + 1) * scale;
    const r32 randf_z = hiprand_uniform(curandstate + idx + 2) * scale;

    value = make_float3(randf_x, randf_y, randf_z);
}

__global__ void setup_cuda_random_state_kernel(hiprandState* curandstate) {
    const i32 idx = threadIdx.x + blockDim.x * blockIdx.x; // subsequence
    const i32 seed = 1234;
    hiprand_init(seed, idx, 0, &curandstate[idx]); // 0 - offset
}


__global__ void normalizeUsingWeightKernel32F(const cv::cuda::PtrStepf weight, 
                                              cv::cuda::PtrStep<sh16> src,
                                              const i32 width, const i32 height)
{
    i32 x = (blockIdx.x * blockDim.x) + threadIdx.x;
    i32 y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < width && y < height)
    {
        constexpr r32 WEIGHT_EPS = 1e-5f;
        const short3 v = ((short3*)src.ptr(y))[x];
        r32 w = weight.ptr(y)[x];
        ((short3*)src.ptr(y))[x] = make_short3(static_cast<sh16>(v.x / (w + WEIGHT_EPS)),
                                               static_cast<sh16>(v.y / (w + WEIGHT_EPS)),
                                               static_cast<sh16>(v.z / (w + WEIGHT_EPS)));
    }
}


__global__ void disparity_to_depth_kernel(const cv::cuda::PtrStepSz<sh16> input_data, r32* output_data,
                                          const ui32 width, const ui32 height,
                                          const r32 focal, const r32 baseline, const r32 depth_scale)
{
    i32 x = (blockIdx.x * blockDim.x) + threadIdx.x;
    i32 y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x >= width && y >= height)
        return;
    
    const sh16 v = ((sh16*)input_data.ptr(y))[x];

    if (v == 0)
        return;
    
    const i32 tid = y * width + x;

    const r32 disp_val = static_cast<r32>(v);
    output_data[tid] = ((focal * baseline) / disp_val) * depth_scale;
}


void normalizeUsingWeightMapGpu32F(const cv::cuda::PtrStepf weight, cv::cuda::PtrStep<sh16> src,
                                   const i32 width, const i32 height)
{
    dim3 blockDim(8, 8);
    dim3 gridDim(divUp(width, blockDim.x), divUp(height, blockDim.y));
    normalizeUsingWeightKernel32F<<<gridDim, blockDim>>> (weight, src, width, height);
}



hipError_t disparity_to_depth(const cv::cuda::PtrStepSz<sh16> input_data, r32* output_data, 
                               const r32 focal, const r32 baseline, const r32 depth_scale) {
    if (!input_data || !output_data)
        return hipErrorInvalidDevicePointer;


    if(input_data.cols == 0 || input_data.rows == 0)
        return hipErrorInvalidValue;
    
    dim3 blockDim(8, 8, 1);
    dim3 gridDim(divUp(input_data.cols, blockDim.x), divUp(input_data.rows, blockDim.y), 1);
    
    disparity_to_depth_kernel<<<gridDim, blockDim, 1>>>(input_data, output_data, 
                                                        input_data.cols, input_data.rows, 
                                                        focal, baseline, depth_scale);
    
    return CUDA(hipGetLastError());
}



hipError_t setup_cuda_rand_state(hiprandState* cu_rand_state) {
    if (!cu_rand_state)
        return hipErrorInvalidDevicePointer;

    setup_cuda_random_state_kernel<<<1, 1>>>(cu_rand_state);

    return CUDA(hipGetLastError());
}

hipError_t generate_random_float3(hiprandState* cu_rand_state, const r32 scale, float3& rand_vec) {
    if (!cu_rand_state)
        return hipErrorInvalidDevicePointer;
    
    rand_vec = make_float3(0, 0, 0);

    random_float3_kernel<<<1, 1>>>(cu_rand_state, scale, rand_vec);
    
    return CUDA(hipGetLastError()); 
}

};
